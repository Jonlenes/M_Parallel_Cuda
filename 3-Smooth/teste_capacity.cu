
#include <hip/hip_runtime.h>
#include <stdio.h>
	
__global__ void kernel(unsigned long long int *count) {
	atomicAdd(count, (unsigned long long int)1);
	//printf("%llu\n", *count);
}


int main() 
{
	unsigned long long int *count;
	unsigned long long int *d_count;
	int size = sizeof(unsigned long long int);
	
	hipMalloc((void **)&d_count, size);
	count = (unsigned long long int *) malloc(size);
	
	dim3 dimBlock (481, 271);//Number of Blocks required
    dim3 dimGrid (32, 32);//Number of threads in each block
	
	hipMemcpy(d_count, 0, size, hipMemcpyHostToDevice);
	kernel <<< dimBlock, dimGrid >>> (d_count); 
		
	hipMemcpy(count, d_count, size, hipMemcpyDeviceToHost);
	
	printf("%llu\n", *count);
	
	free(count);
	hipFree(d_count);
	
	return 0;
}
